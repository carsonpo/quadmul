#include <hip/hip_runtime.h>
#include <cuda_pipeline.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_fp16.h"

#define CUDA_DEVICE_INLINE __device__ __forceinline__

typedef struct
{
    int x;
} Coord1D;

typedef struct
{
    int x, y;
} Coord2D;

typedef struct
{
    int x, y, z;
} Coord3D;

typedef struct
{
    int x, y, z, t;
} Coord4D;

typedef struct
{
    int x, y, z, t, u;
} Coord5D;

template <typename T, int Dims, typename Derived>
class BaseTensor
{
public:
    CUDA_DEVICE_INLINE
    BaseTensor(void *ptr)
        : startPtr(reinterpret_cast<T *>(ptr)), endPtr(reinterpret_cast<T *>(ptr) + static_cast<const Derived *>(this)->totalSize()) {}

    T *startPtr;
    T *endPtr;

    CUDA_DEVICE_INLINE T get() const { return startPtr[0]; }
    CUDA_DEVICE_INLINE T get(int x) const { return startPtr[x]; }
    CUDA_DEVICE_INLINE T get(int x, int y) const { return startPtr[x * static_cast<const Derived *>(this)->stride().x + y]; }
    CUDA_DEVICE_INLINE T get(int x, int y, int z) const { return startPtr[x * static_cast<const Derived *>(this)->stride().x + y * static_cast<const Derived *>(this)->stride().y + z]; }

    CUDA_DEVICE_INLINE void set(const T value) { startPtr[0] = value; }
    CUDA_DEVICE_INLINE void set(int x, const T value) { startPtr[x] = value; }
    CUDA_DEVICE_INLINE void set(int x, int y, const T value) { startPtr[x * static_cast<const Derived *>(this)->stride().x + y] = value; }
    CUDA_DEVICE_INLINE void set(int x, int y, int z, const T value) { startPtr[x * static_cast<const Derived *>(this)->stride().x + y * static_cast<const Derived *>(this)->stride().y + z] = value; }

    CUDA_DEVICE_INLINE T *get_ptr() { return startPtr; }
    CUDA_DEVICE_INLINE T *get_ptr(int x) { return &startPtr[x]; }
    CUDA_DEVICE_INLINE T *get_ptr(int x, int y) { return &startPtr[x * static_cast<const Derived *>(this)->stride().x + y]; }
    CUDA_DEVICE_INLINE T *get_ptr(int x, int y, int z) { return &startPtr[x * static_cast<const Derived *>(this)->stride().x + y * static_cast<const Derived *>(this)->stride().y + z]; }

    template <typename U>
    CUDA_DEVICE_INLINE U get_reinterpreted() const
    {
        return *reinterpret_cast<const U *>(startPtr);
    }

    template <typename U>
    CUDA_DEVICE_INLINE U get_reinterpreted(int x) const
    {
        return reinterpret_cast<const U *>(startPtr)[x];
    }

    template <typename U>
    CUDA_DEVICE_INLINE U get_reinterpreted(int x, int y) const
    {
        return reinterpret_cast<const U *>(startPtr)[(x * static_cast<const Derived *>(this)->stride().x + y) * sizeof(T) / sizeof(U)];
    }

    template <typename U>
    CUDA_DEVICE_INLINE U get_reinterpreted(int x, int y, int z) const
    {
        return reinterpret_cast<const U *>(startPtr)[(x * static_cast<const Derived *>(this)->stride().x + y * static_cast<const Derived *>(this)->stride().y + z) * sizeof(T) / sizeof(U)];
    }

    template <typename U>
    CUDA_DEVICE_INLINE void set_reinterpreted(U value)
    {
        *reinterpret_cast<U *>(startPtr) = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE void set_reinterpreted(int x, U value)
    {
        reinterpret_cast<U *>(startPtr)[x] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE void set_reinterpreted(int x, int y, U value)
    {
        reinterpret_cast<U *>(startPtr)[(x * static_cast<const Derived *>(this)->stride().x + y) * sizeof(T) / sizeof(U)] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE void set_reinterpreted(int x, int y, int z, U value)
    {
        reinterpret_cast<U *>(startPtr)[(x * static_cast<const Derived *>(this)->stride().x + y * static_cast<const Derived *>(this)->stride().y + z) * sizeof(T) / sizeof(U)] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE U *get_ptr_reinterpreted()
    {
        return reinterpret_cast<U *>(startPtr);
    }

    template <typename U>
    CUDA_DEVICE_INLINE U *get_ptr_reinterpreted(int x)
    {
        return &reinterpret_cast<U *>(startPtr)[x];
    }

    template <typename U>
    CUDA_DEVICE_INLINE U *get_ptr_reinterpreted(int x, int y)
    {
        return &reinterpret_cast<U *>(startPtr)[(x * static_cast<const Derived *>(this)->stride().x + y) * sizeof(T) / sizeof(U)];
    }

    template <typename U>
    CUDA_DEVICE_INLINE U *get_ptr_reinterpreted(int x, int y, int z)
    {
        return &reinterpret_cast<U *>(startPtr)[(x * static_cast<const Derived *>(this)->stride().x + y * static_cast<const Derived *>(this)->stride().y + z) * sizeof(T) / sizeof(U)];
    }
};

template <typename T>
class SmemTensor0D : public BaseTensor<T, 0, SmemTensor0D<T>>
{
public:
    using BaseTensor<T, 0, SmemTensor0D<T>>::BaseTensor;
    CUDA_DEVICE_INLINE int totalSize() const { return 1; }
};

template <typename T, int ShapeX>
class SmemTensor1D : public BaseTensor<T, 1, SmemTensor1D<T, ShapeX>>
{
public:
    using BaseTensor<T, 1, SmemTensor1D<T, ShapeX>>::BaseTensor;
    CUDA_DEVICE_INLINE Coord1D shape() const { return {ShapeX}; }
    CUDA_DEVICE_INLINE int totalSize() const { return ShapeX; }
};

template <typename T, int ShapeX, int ShapeY>
class SmemTensor2D : public BaseTensor<T, 2, SmemTensor2D<T, ShapeX, ShapeY>>
{
public:
    using BaseTensor<T, 2, SmemTensor2D<T, ShapeX, ShapeY>>::BaseTensor;
    CUDA_DEVICE_INLINE Coord2D shape() const { return {ShapeX, ShapeY}; }
    CUDA_DEVICE_INLINE Coord1D stride() const { return {ShapeY}; }
    CUDA_DEVICE_INLINE int totalSize() const { return ShapeX * ShapeY; }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeY> get_child(int x)
    {
        return SmemTensor1D<T, ShapeY>(this->startPtr + x * this->stride().x);
    }
};

template <typename T, int ShapeX, int ShapeY, int ShapeZ>
class SmemTensor3D : public BaseTensor<T, 3, SmemTensor3D<T, ShapeX, ShapeY, ShapeZ>>
{
public:
    using BaseTensor<T, 3, SmemTensor3D<T, ShapeX, ShapeY, ShapeZ>>::BaseTensor;
    CUDA_DEVICE_INLINE Coord3D shape() const { return {ShapeX, ShapeY, ShapeZ}; }
    CUDA_DEVICE_INLINE Coord2D stride() const { return {ShapeY * ShapeZ, ShapeZ}; }
    CUDA_DEVICE_INLINE int totalSize() const { return ShapeX * ShapeY * ShapeZ; }

    CUDA_DEVICE_INLINE
    SmemTensor2D<T, ShapeY, ShapeZ> get_child(int x)
    {
        return SmemTensor2D<T, ShapeY, ShapeZ>(this->startPtr + x * this->stride().x);
    }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeZ> get_child(int x, int y)
    {
        return SmemTensor1D<T, ShapeZ>(this->startPtr + x * this->stride().x + y * this->stride().y);
    }
};

template <typename T>
class GMemTensor0D : public BaseTensor<T, 0, GMemTensor0D<T>>
{
public:
    using BaseTensor<T, 0, GMemTensor0D<T>>::BaseTensor;
    CUDA_DEVICE_INLINE int totalSize() const { return 1; }
};

template <typename T>
class GMemTensor1D : public BaseTensor<T, 1, GMemTensor1D<T>>
{
private:
    int shapeX;

public:
    CUDA_DEVICE_INLINE
    GMemTensor1D(void *gmemPtr, int x) : BaseTensor<T, 1, GMemTensor1D<T>>(gmemPtr), shapeX(x) {}

    CUDA_DEVICE_INLINE Coord1D shape() const { return {shapeX}; }
    CUDA_DEVICE_INLINE int totalSize() const { return shapeX; }
};

template <typename T>
class GMemTensor2D : public BaseTensor<T, 2, GMemTensor2D<T>>
{
private:
    int shapeX, shapeY;

public:
    CUDA_DEVICE_INLINE
    GMemTensor2D(void *gmemPtr, int x, int y) : BaseTensor<T, 2, GMemTensor2D<T>>(gmemPtr), shapeX(x), shapeY(y) {}

    CUDA_DEVICE_INLINE Coord2D shape() const { return {shapeX, shapeY}; }
    CUDA_DEVICE_INLINE Coord1D stride() const { return {shapeY}; }
    CUDA_DEVICE_INLINE int totalSize() const { return shapeX * shapeY; }

    CUDA_DEVICE_INLINE
    GMemTensor1D<T> get_child(int x)
    {
        return GMemTensor1D<T>(this->startPtr + x * this->stride().x, shapeY);
    }
};

template <typename T>
class GMemTensor3D : public BaseTensor<T, 3, GMemTensor3D<T>>
{
private:
    int shapeX, shapeY, shapeZ;

public:
    CUDA_DEVICE_INLINE
    GMemTensor3D(void *gmemPtr, int x, int y, int z) : BaseTensor<T, 3, GMemTensor3D<T>>(gmemPtr), shapeX(x), shapeY(y), shapeZ(z) {}

    CUDA_DEVICE_INLINE Coord3D shape() const { return {shapeX, shapeY, shapeZ}; }
    CUDA_DEVICE_INLINE Coord2D stride() const { return {shapeY * shapeZ, shapeZ}; }
    CUDA_DEVICE_INLINE int totalSize() const { return shapeX * shapeY * shapeZ; }

    CUDA_DEVICE_INLINE
    GMemTensor2D<T> get_child(int x)
    {
        return GMemTensor2D<T>(this->startPtr + x * this->stride().x, shapeY, shapeZ);
    }

    CUDA_DEVICE_INLINE
    GMemTensor1D<T> get_child(int x, int y)
    {
        return GMemTensor1D<T>(this->startPtr + x * this->stride().x + y * this->stride().y, shapeZ);
    }
};